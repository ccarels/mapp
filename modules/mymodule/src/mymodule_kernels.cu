#include "hip/hip_runtime.h"
#include "mymodule_kernels.hpp"

#include <stdio.h>

/**
 * \brief A basic CUDA kernel.
 */
__global__ void mymodule_kernel() {
    printf("Hello world from CUDA kernel from thread %d\n", threadIdx.x);
}

void mymodule_kernel_wrapper() {
    printf("Will call kernel\n");
    mymodule_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    printf("Done calling kernel\n");
}
